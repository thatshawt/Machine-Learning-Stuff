#include "hip/hip_runtime.h"
#pragma once

#include "SpikingNeurons.h"
#include "helper.h"
#include "hip/hip_runtime.h"
#include ""
#include <thrust/device_vector.h>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
#define gpuLog(ans) { gpuAssert((ans), __FILE__, __LINE__, false); }
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) {
			getchar();
			exit(code);
		}
	}
}

NeuronParams NeuronParams::random() {
	NeuronParams params;
	params.inhibit = !(frand(1) <= 0.8);

	if (params.inhibit) {
		//this->weights[j][i] = -frand(1);
		params.a = 0.02f + 0.08f * frand(1);
		params.b = 0.25f - 0.05f * frand(1);
		params.c = -65;
		params.d = 2;
	} else {
		//this->weights[j][i] = 0.5f * frand(1);
		params.a = 0.02f;
		params.b = 0.2f;
		params.c = -65 + 15 * pow(frand(1), 2);
		params.d = 8 - 6 * pow(frand(1), 2);
	}
	return params;
}


SpikingNeuron::SpikingNeuron() {

}

SpikingNeuron::SpikingNeuron(NeuronParams params) {
	this->params = params;

	this->v = -65;
	this->u = this->params.b * this->v;
}

MaskConfiguration MaskConfiguration::getDefault(int neurons) {
	MaskConfiguration config;
	config.start = 0;
	//config.length = -1;
	for (int i = 0; i < neurons; i++) {
		config.stdpMask.push_back(std::vector<char>(neurons, true));
		config.weightMask.push_back(std::vector<char>(neurons, true));
	}
	return config;
}



SNN::SNN(int neurons, std::vector<NeuronParams> neuronConfigs) {
	srand(std::time(0));

	this->neurons.reserve(neurons);
	weights.reserve(neurons);
	//weightMask.reserve(neurons);
	//stdpMask.reserve(neurons);
	preSynapticTiming.reserve(neurons);
	postSynapticTiming.reserve(neurons);
	for (int i = 0; i < neurons; i++) {
		//printf("i: %d\n",i);
		this->neurons.push_back(struct SpikingNeuron());
		weights.push_back(std::vector<float>(neurons,0));
		//weightMask.push_back(std::vector<bool>(neurons,true));
		//stdpMask.push_back(std::vector<bool>(neurons,true));
		preSynapticTiming.push_back(std::vector<int>(neurons,-1));
		postSynapticTiming.push_back(std::vector<int>(neurons, -1));
	}

	for (int i = 0; i < neurons;i++) {
		for (int j = 0; j < neurons; j++) {
			SpikingNeuron neuron = this->neurons[j];
			neuron = SpikingNeuron(neuronConfigs[j]);
				
			if (neuron.params.inhibit) {
				this->weights[j][i] = -frand(1);
			}
			else {
				this->weights[j][i] = 0.5f * frand(1);
			}
		}
	}
		
	for (int i = 0; i < neurons && false; i++) {
		//weights[i].reserve(neurons);
		//weightMask[i].reserve(neurons);
		preSynapticTiming[i].reserve(neurons);
		postSynapticTiming[i].reserve(neurons);
			
		//fill_n(weightMask[i], true);
		fill_n(postSynapticTiming[i], -1);
		fill_n(preSynapticTiming[i], -1);
	}
}
void SNN::addMaskConfig(MaskConfiguration phase) {
	this->maskConfigs.push_back(phase);
}

void SNN::tickWithSTDP(unsigned long long milliseconds, bool debug, float(*getInput)(int)) {
	int weightChanges = 0;
	int currentPhase = 0;
	unsigned long long prevT = t;
	for (;t-prevT < milliseconds; t++) {
		for (int j = 0; j < neurons.size(); j++) {
			//these are pointer to the masks we are gonna use
			std::vector<std::vector<char>>* stdpMask = &maskConfigs[currentPhase].stdpMask;
			std::vector<std::vector<char>>* weightMask = &maskConfigs[currentPhase].weightMask;

			SpikingNeuron *firingNeuron = &neurons[j];
			//firingNeuron->I = !firingNeuron->inhibit ? 5 * frand(1) : 2 * frand(1);//thalamic input
			firingNeuron->I = getInput(j);
			//printf("ratto: %f\n", firingNeuron->I);
			if (firingNeuron->v >= 30) {//it fired
				//printf("fired\n");
				//we need to do, timeDifference = timePost - timePre, then plug difference into the STDP learning equations

				firingNeuron->v = firingNeuron->params.c;
				firingNeuron->u += firingNeuron->params.d;

				// increment each I by the weights of the fired neurons
				// I = I + sum(S(:, fired);
				firingNeuron->I += 1;

				for (int i = 0; i < weights.size(); i++) {
					if ((*weightMask)[i][j])postSynapticTiming[i][j] = t;
				}
				for (int i = 0; i < weights[j].size(); i++) {//loops through the post neuron given the pre neuron
					if ((*weightMask)[j][i]) {
						preSynapticTiming[j][i] = t;

						SpikingNeuron *postSynaptic = &neurons[i];
						postSynaptic->I += weights[j][i];//send the signal to other neurons
						//check the timings to get the dt

						if ((*stdpMask)[j][i] && preSynapticTiming[j][i] != -1 && postSynapticTiming[j][i] != -1) {
							//courtesy of http://www.scholarpedia.org/article/Spike-timing_dependent_plasticity
							int dt = postSynapticTiming[j][i] - preSynapticTiming[j][i];
							float dw;
								
							if (dt > 0) {
								dw = (float)(aPlus * exp((-1 * dt) / tauPlus));//these are popular equations apparently
							}
							else {
								dw = (float)(-1 * aMinus * exp(dt / tauMinus));//this one as WELL
							}

							if (debug && dw > 0.01) printf("%d learn by this mushc: %f",j,dw);
							weights[j][i] += dw;
							preSynapticTiming[j][i] = -1;
							postSynapticTiming[j][i] = -1;
							weightChanges++;
						}
					}
				}
			}
			else {
				firingNeuron->v += (0.04 * pow(firingNeuron->v, 2) + 5 * (double)firingNeuron->v + 140 - firingNeuron->u + firingNeuron->I);
				//firingNeuron.v += 0.5*(0.04*Math.pow(firingNeuron.v,2) + 5*firingNeuron.v + 140 - firingNeuron.u + firingNeuron.I);
				//firingNeuron.v += 0.5*(0.04*Math.pow(firingNeuron.v,2) + 5*firingNeuron.v + 140 - firingNeuron.u + firingNeuron.I);
				firingNeuron->u += firingNeuron->params.a * (firingNeuron->params.b * firingNeuron->v - firingNeuron->u);
			}
			if (debug) printf("%d weight changes\n", weightChanges);
			for (int phase = 0; phase < maskConfigs.size();phase++) {
				if (maskConfigs[phase].start == t) {
					currentPhase = phase;
					break;
				}
			}
		}
	}
}

#define T 1024

__global__
void neuronKernel(SpikingNeuron* neurons, int length) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < length) {
		SpikingNeuron neuron = neurons[i];
		if (neuron.v >= 30) {
			neuron.v = neuron.params.c;
			neuron.u += neuron.params.d;
			//TODO: implement the I thing
		}
		neuron.v += (0.04 * (neuron.v * neuron.v) + 5 * (double)neuron.v + 140 - neuron.u + neuron.I);
		neuron.u += neuron.params.a * (neuron.params.b * neuron.v - neuron.u);
	}

}

void SNN::tickImproved(long milliseconds) {
	int weightChanges = 0;
	int currentPhase = 0;
	unsigned long long prevT = t;

	hipMallocManaged((void**)&neurons[0], sizeof(SpikingNeuron) * neurons.size());
	hipMallocManaged((void**)&postSynapticTiming[0], sizeof(postSynapticTiming) * postSynapticTiming.size());

	for (; t - prevT < milliseconds; t++) {
		std::vector<std::vector<char>>* stdpMask = &maskConfigs[currentPhase].stdpMask;
		std::vector<std::vector<char>>* weightMask = &maskConfigs[currentPhase].weightMask;

		neuronKernel <<<(int)ceil(neurons.size() / T), T >>> (&neurons[0], neurons.size());

		hipDeviceSynchronize();

		//MaskConfig check
		for (int phase = 0; phase < maskConfigs.size(); phase++) {
			if (maskConfigs[phase].start == t) {
				currentPhase = phase;
				break;
			}
		}
	}

	hipFree(&neurons);
}

int SNN::pruneWeakConnections(int phase, float differenceThreshold, bool debug) {
	int pruned = 0;
	for (int j = 0; j < neurons.size(); j++) {
		for (int i = 0; i < neurons.size(); i++) {
			if (maskConfigs[phase].weightMask[j][i] && abs(weights[j][i]) <= abs(differenceThreshold)) {
				maskConfigs[phase].weightMask[j][i] = false;
				pruned++;
			}
		}
	}
	if (debug) printf("%d pruned\n",pruned);
	return pruned;
}

__global__
void pruningKernel(int* weightRows, int* weightMaskRows,
					int col, int row, float threshold) {
	//printf("blockIdx: %d, blockDimx: %d, threadIdx: %d\n", blockIdx.x, blockDim.x, threadIdx.x);
	int i = (blockIdx.x * blockDim.x) + threadIdx.x;
	//printf("(%d)",i);
	if (i < col*row && i >= 0) {
		int x = (int)ceilf((float)(i/col));
		int y = i % col;

		printf("%d ", i); //TODO: this code block randomly gets executed what

		bool weightMask = *((int*)*(weightMaskRows + y) + x);
		float weight = *((int*)*(weightRows + y) + x);
		printf("weight: %0.2f, ", weight);//TODO: this isnt printing anything bruh
		if (weightMask && abs(weight) <= abs(threshold)) {
			*((int*)*(weightRows + y) + x) = false;
			printf("1");
		}
	}
}

int SNN::pruneWeakConnectionsImproved(int phase, int threshold){
	int before = getActiveConnections(phase);

	int* weightRows = new int[weights.size()];
	int* weightMaskRows = new int[maskConfigs[phase].weightMask.size()];

	for (int i = 0; i < weights.size();i++) {
		//printf("create row %d\n",i);
		//weightRows[i] = (float*)malloc(sizeof(float));
		weightRows[i] = (int)weights[i].data();
	}
	for (int i = 0; i < maskConfigs[phase].weightMask.size();i++) {
		//printf("create row %d\n", i);
		//weightMaskRows[i] = (char*)malloc(sizeof(char));
		weightMaskRows[i] = (int)maskConfigs[phase].weightMask[i].data();
	}

	printf("mask: %d, weights; %d, weightRows: %d, weightMaskRows: %d\n",
		sizeof(MaskConfiguration) * maskConfigs.size(),
		sizeof(float) * weights.size() * weights[0].size(),
		weights.size() * sizeof(float*),
		maskConfigs[phase].weightMask.size() * sizeof(char*));

	size_t free, total;
	gpuLog(hipMemGetInfo(&free, &total));
	//printf("usage: %d, free: %d\n", total-free, free);

	//printf("active connetions: %d\n", getActiveConnections(phase));
	gpuErrchk(hipMallocManaged((void**)weightRows, weights.size() * sizeof(int)));
	gpuLog(hipMemGetInfo(&free, &total));
	//printf("usage: %d, free: %d\n", total - free, free);
	gpuErrchk(hipMallocManaged((void**)weightMaskRows, maskConfigs[phase].weightMask.size() * sizeof(int)));
	gpuLog(hipMemGetInfo(&free, &total));
	//printf("usage: %d, free: %d\n", total - free, free);

	//printf("active connetions: %d\n", getActiveConnections(phase));
	gpuErrchk(hipMallocManaged((void**)&maskConfigs, sizeof(MaskConfiguration) * maskConfigs.size()));
	gpuLog(hipMemGetInfo(&free, &total));
	//printf("usage: %d, free: %d\n", total - free, free);
	gpuErrchk(hipMallocManaged((void**)&weights, sizeof(float) * weights.size()* weights[0].size()));
	gpuLog(hipMemGetInfo(&free, &total));
	//printf("usage: %d, free: %d\n", total - free, free);

	//printf("active connetions: %d\n", getActiveConnections(phase));

	int blocks = (int)ceil((double)weights.size() / (double)T);
	//printf("blocks: %d\n",blocks);
	pruningKernel<<<blocks, T>>>(//&(maskConfigs[phase].weightMask),
														//&weights,
															weightRows, weightMaskRows,
															weights[0].size(), weights.size(), threshold);
	hipDeviceSynchronize();

	printf("active connetions: %d\n",getActiveConnections(phase));

	hipFree(&maskConfigs);
	hipFree(&weights);
	hipFree(weightRows);
	hipFree(weightMaskRows);

	delete[] weightRows;
	delete[] weightMaskRows;

	printf("active connetions: %d\n", getActiveConnections(phase));

	return before - getActiveConnections(phase);
}

SpikingNeuron SNN::getNeuron(int i) {
	return neurons[i];
}

int SNN::getActiveConnections(int phase) {
	int count = 0;
	for (int i = 0; i < maskConfigs[phase].weightMask.size(); i++) {
		for (int j = 0; j < maskConfigs[phase].weightMask[i].size(); j++) {
			if (maskConfigs[phase].weightMask[i][j])count++;
		}
	}
	return count;
}

void SNN::printNeuron(SpikingNeuron *neuron) {
	printf("v:%.1f u:%.1f I:%.1f\n",neuron->v, neuron->u, neuron->I);
}

void SNN::printNeurons() {
	for (int i = 0; i < neurons.size();i++) {
		printf("%d ",i);
		printNeuron(&neurons[i]);
	}
}

void SNN::doNeuronRat(SpikingNeuron* neurons, int length) {
}