#include "hip/hip_runtime.h"
#pragma once

//#include "hip/hip_runtime.h"
//#include <>

#include <stdio.h>
#include "../Machine Learning Stuff/SpikingNeurons.cpp"

//__global__ void firedNeuronKernel(SpikingNeuron* neurons, int length);
__global__ void firedNeuronKernel(SpikingNeuron* neurons, int length) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < length) {
        SpikingNeuron neuron = neurons[i];
        
    	//    v(fired)=c(fired);
		//    u(fired)=u(fired)+d(fired);
		//    I=I+sum(S(:,fired),2);
        
        if (neuron.v >= 30) {
            neuron.v = neuron.params.c;
            neuron.u += neuron.params.d;
            //TODO: implement the "I=I+sum(S(:,fired),2);"
        }
        neuron.v += (0.04 * (neuron.v * neuron.v) + 5 * (double)neuron.v + 140 - neuron.u + neuron.I);
        neuron.u += neuron.params.a * (neuron.params.b * neuron.v - neuron.u);
    }

}

#define T 1024
void doNeuronShit(SpikingNeuron* neurons, int length) {
    hipMallocManaged(&neurons, sizeof(SpikingNeuron)*length);
    
    firedNeuronKernel <<<(int)ceil(length / T), T >>> (neurons, length);
    hipDeviceSynchronize();

    //neuronKernel<<<(int)ceil(length / T), T >>> (neurons, length);
    //hipDeviceSynchronize();
    
    hipFree(&neurons);
}
