#include "hip/hip_runtime.h"
#pragma once
#define T 1024

#include "epic.h"
#include "SpikingNeurons.h"
#include "hip/hip_runtime.h"
#include ""

__global__ void epic::neuronKernel(SpikingNeuron* neurons, int length) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < length) {
		SpikingNeuron neuron = neurons[i];
		if (neuron.v >= 30) {
			neuron.v = neuron.params.c;
			neuron.u += neuron.params.d;
			//TODO: implement the I thing
		}
		neuron.v += (0.04 * (neuron.v * neuron.v) + 5 * (double)neuron.v + 140 - neuron.u + neuron.I);
		neuron.u += neuron.params.a * (neuron.params.b * neuron.v - neuron.u);
	}

}

void epic::doNeuronRat(SpikingNeuron* neurons, int length) {
	hipMallocManaged((void**)neurons, sizeof(SpikingNeuron) * length);

	neuronKernel <<<(int)ceil(length / T), T >>> (neurons, length);
	hipDeviceSynchronize();

	hipFree(&neurons);
}