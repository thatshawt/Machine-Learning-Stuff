#include "hip/hip_runtime.h"
#include <stdio.h>
#include <vector>
#include "../Common/helper.h"
#include <thrust\host_vector.h>
#include <thrust\device_vector.h>

using namespace std;
using namespace thrust;

template <class size_t>
class Cuda2D {
private:
	uintptr_t* rowStart;
	int rowSize;
	vector<uintptr_t> rows;
	vector<std::vector<size_t>>* data;

	void updateRows() {
		rows.clear();
		for (int r = 0; r < data->size(); r++) {
			rows.push_back((uintptr_t)(data->at(r).data()));
			printf("r: %d, ", r);
			printf("row size: %d, ",rows.size());
			printf("val: %p, ", rows[r]);
		}
		printf("\n");
		/*
		printf("\n0,0: %p unsigned cast data[]\n", (uint64_t)data[0].data());
		printf("0,0: %p unsigned cast data->at()\n", (uint64_t)data->at(0).data());
		printf("0,0: %p data->at()\n", data->at(0).data());
		printf("0,0: %p data[]\n", data[0].data());
		printf("0,0: %p unsigned cast data.data()\n", (uint64_t)data->data());
		printf("0,0: %p data.data()\n", data->data());
		*/
		rowSize = rows.size();
		rowStart = rows.data();
	}
public:
	std::vector<size_t>* getRow(int r) {
		return &data->at(r);
	}

	void setData(std::vector<std::vector<size_t>>* bruh) {
		this->data = bruh;
		updateRows();
	}

	__forceinline__ __host__ __device__ size_t* safeAt(int r, int i) {
		if (r + 1 > rowSize) {
			printf("too big\n");
			return nullptr;
		}
		printf("wtf is happening rn bas\n");
		//   *(int*)(void*)(uintptr_t)(void*)ints.at(0).data();
		return ((size_t*)(*(rowStart + r)) + i);
	}
};
__global__
void doFunction(Cuda2D<int>* bruh) {
	printf("bruh");
	printf("size: %d\n", bruh->safeAt(0,0));
}

int main() {
	std::vector<int> bruhs;
	bruhs.push_back(1);
	bruhs.push_back(2);
	bruhs.push_back(3);

	int i = 0;
	printf("%d: %d\n",i, *(bruhs.data()+i));

	Cuda2D<int> v2;

	std::vector<std::vector<int>> ints;
	ints.push_back({1,2,3});
	ints.push_back({4,5,6});

	void* originalPtr = ints.at(0).data();
	uintptr_t theIntVersion = (uintptr_t)originalPtr;
	void* newPtr = (void*)theIntVersion;

	int allInOne = *(int*)(void*)(uintptr_t)(void*)ints.at(0).data();

	printf("0,1: %d\n", *(int*)originalPtr);
	printf("0,1: %d\n", *(int*)newPtr);
	printf("0,1: %d\n", allInOne);

	v2.setData(&ints);

	printf("0,1: %d\n", v2.safeAt(0,0));

	int intsSize = ints.size();

	hipMallocManaged((void**)&v2, sizeof(Cuda2D<int>));
	hipMallocManaged((void**)&ints, sizeof(std::vector<int>) * ints.size());
	//ints.size() changes after doing this for some reason

	for (int r = 0; r < intsSize; r++) {
		std::vector<int>* row = v2.getRow(r);
		printf("ints size: %p, row: %d, row size: %d\n", ints.size(), r, row->size());
		hipMallocManaged((void**)row, row->size() * sizeof(int));
	}

	printf("ints size: %d\n", ints.size());

	doFunction <<<1, 1 >>> (&v2);

	hipDeviceSynchronize();

	hipFree(&v2);
	hipFree(&ints);

	for (int r = 0; r < intsSize; r++) {
		std::vector<int>* row = v2.getRow(r);
		hipFree(row);
	}

	getchar();

	return 1;
}